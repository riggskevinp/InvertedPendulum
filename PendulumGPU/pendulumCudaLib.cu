#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <chrono>

#include "pendulumCudaLib.cuh"

template<class F>
__global__ void exKernel(F f,double *device_a, double *device_b, double time_change){
    int threadId = threadIdx.x + blockIdx.x * blockDim.x ;
    f(device_a, device_b, time_change, threadId);
}

template<typename F>
void PendulumCudaLib::executeArrayFunction(F fn, double* state, double* stateChange, double time_change, int size)
{
    const int numThreads = 4;
    const int blockSize = 4;
    const int numBlocks = numThreads/blockSize;
    double *host_state, *host_stateChange;
    double *device_state, *device_stateChange;
    //int sizeOfArray = 1024*1024;

    hipMalloc(( void**)& device_state, numThreads * sizeof(double));
    hipMalloc(( void**)& device_stateChange, numThreads * sizeof(double));


    hipHostAlloc((void **)&host_state, numThreads * sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void **)&host_stateChange, numThreads * sizeof(double), hipHostMallocDefault);


    for(int index = 0; index < size; index++)
    {
        host_state[index] = state[index];
        host_stateChange[index] = stateChange[index];
    }

    hipMemcpy(device_state, host_state, numThreads * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(device_stateChange, host_stateChange, numThreads * sizeof(double), hipMemcpyHostToDevice);

    /*Kernel call*/

    exKernel<<<1, 4>>>(fn, device_state, device_stateChange, time_change);

    hipMemcpy(host_state, device_state, numThreads * sizeof(double), hipMemcpyDeviceToHost);

    for(int index = 0; index < size; index++)
    {
        state[index] = host_state[index];
    }

    hipHostFree(host_state);
    hipHostFree(host_stateChange);
    hipFree(device_state);
    hipFree(device_stateChange);
}


void PendulumCudaLib::updateState(double* state, double* state_change, double time_change)
{
    auto updateState = [] __device__ (double* state, double* state_change, double time_change, int index){
        state[index]= state[index] + state_change[index] * 0.0001;
    };
    PendulumCudaLib::executeArrayFunction(updateState, state, state_change, time_change, 4);
}
